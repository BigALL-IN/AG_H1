#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <vector>
#include <cmath>
#include <random>
#include <iostream>
#include <numeric>
#include <iomanip>

#include "kernel.cuh"


__constant__ Config globalConfig;


__global__ void Init_states(hiprandState* states, long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    hiprand_init(seed, idx, 1000, &states[idx]);
}

__global__ void InitBitstring(hiprandState* states, bool* b) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    int startBit = idx * globalConfig.bits;
    for (int i = startBit; i < startBit + globalConfig.bits; i++)
    {
        b[i] = hiprand_uniform(&states[idx]) > 0.5f;
    }
}

__device__ void Convert(bool* bits, double* values)
{
    for (int j = 0; j < globalConfig.d; j++) {
        unsigned long long dec = 0;
        for (int i = 0; i < globalConfig.bitsPerDim; i++)
        {
            dec = (dec << 1) | bits[j * globalConfig.bitsPerDim + i];

        }
        values[j] = globalConfig.a + dec * (globalConfig.b - globalConfig.a) / ((1ull << globalConfig.bitsPerDim) - 1);
    }
}
__global__ void GenRealValues(bool* bits, double* values) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    Convert(bits + idx * globalConfig.bits, values + idx * globalConfig.d);
}

__device__ double Rastrigin(double* v, int dimensions) {

    double res = 10 * dimensions;
    for (int i = 0; i < dimensions; i++) {
        res += v[i] * v[i] - 10 * cos(2 * M_PI * v[i]);
    }
    return res;
}

__device__ double Michalewicz(double* v, int dimensions) {
    double res = 0;
    for (int i = 0; i < dimensions; i++) {
        res += sin(v[i]) * pow(sin(((i + 1) * v[i] * v[i]) / M_PI), 20);
    }
    return -res;
}

//reminder to check if this si actually dejong
__device__ double Dejong(double* v, int dimensions) {
    double res = 0;
    for (int i = 0; i < dimensions; i++) {
        res += v[i] * v[i];
    }
    return res;
}

__device__ double Schwefel(double* v, int dimensions) {
    double res = 0;
    for (int i = 0; i < dimensions; i++) {
        res += -v[i]*sin(sqrt(abs(v[i])));
    }
    return res;
}

__device__ double Eval(double* values)
{
    switch (globalConfig.func)
    {
    case function::Rastrigin:
        return Rastrigin(values, globalConfig.d);
        break;
    case function::Michalewicz:
        return Michalewicz(values, globalConfig.d);
        break;

    case function::Schwefel:
        return Schwefel(values, globalConfig.d);
        break;

    case function::Dejong:
        return Dejong(values, globalConfig.d);
        break;
    }
}

__global__ void EvalFitness(double* values, double* candidates)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    candidates[idx] = Eval(idx * globalConfig.d + values);
}


__global__  void HillClimbFirstImpr(bool* bitstr, double* values, double* candidates) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    int startBit = idx * globalConfig.bits;
    double bestValue = candidates[idx];
    double currentValue = bestValue;

    for (int i = 0; i < globalConfig.bits; i++)
    {
        int bitflip = startBit + i;
        bitstr[bitflip] = !bitstr[bitflip];
        Convert(bitstr + startBit, values + idx * globalConfig.d);
        currentValue = Eval(values + idx * globalConfig.d);

        if (currentValue < bestValue)
        {
            bestValue = currentValue;
           
            i = 0;
        }
        else { bitstr[bitflip] = !bitstr[bitflip]; }
    }

    candidates[idx] = bestValue;
}

__global__  void HillClimbBestImpr(bool* bitstr, double* values, double* candidates) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    int startBit = idx * globalConfig.bits;
    int bestbit = 0;
    double bestValue = candidates[idx];
    double currentValue = bestValue;
    bool improved = 1;
    while (improved) {
        improved = 0;
        for (int i = 0; i < globalConfig.bits; i++)
        {
            int bitflip = startBit + i;
            bitstr[bitflip] = !bitstr[bitflip];
            Convert(bitstr + startBit, values + idx * globalConfig.d);
            currentValue = Eval(values + idx * globalConfig.d);

            if (currentValue < bestValue)
            {
                bestValue = currentValue;
                bestbit = bitflip;
                improved = 1;
            }
            bitstr[bitflip] = !bitstr[bitflip];
        }
        if (improved) {
            bitstr[bestbit] = !bitstr[bestbit];
        }

    }

    candidates[idx] = bestValue;
}
__global__  void HillClimbWorstImpr(bool* bitstr, double* values, double* candidates) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    int startBit = idx * globalConfig.bits;
    int bestbit = 0;
    double bestValue = candidates[idx];
    double currentValue = bestValue;
    bool improved = 1;
    while (improved) {
        improved = 0;
        for (int i = 0; i < globalConfig.bits; i++)
        {
            int bitflip = startBit + i;
            bitstr[bitflip] = !bitstr[bitflip];
            Convert(bitstr + startBit, values + idx * globalConfig.d);
            currentValue = Eval(values + idx * globalConfig.d);
            double initValue = currentValue;
            if ((currentValue < bestValue) && (!improved))
            {
                bestValue = currentValue;
                bestbit = bitflip;
                improved = 1;
            } else if ((currentValue > bestValue) && (currentValue < initValue))
            {
                bestValue = currentValue;
                bestbit = bitflip;
            }
              
            bitstr[bitflip] = !bitstr[bitflip];
        }
        if (improved) {
            bitstr[bestbit] = !bitstr[bestbit];
        }

    }

    candidates[idx] = bestValue;
}
__global__  void Annealing(bool* bitstr, double* values, double* candidates, hiprandState* states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= globalConfig.it) return;
    int startBit = idx * globalConfig.bits;
    int T = 1000 * pow(0.95, idx);
    int bestbit = 0;
    double bestValue = candidates[idx];
    double currentValue = bestValue;
    int counter = 0;
    int changeCount = 0;
    int maxAttempts = 100;

    do {
        
        for (int i = 0; i < globalConfig.bits; i++)
        {
            int bitflip = startBit + i;
            bitstr[bitflip] = !bitstr[bitflip];
            Convert(bitstr + startBit, values + idx * globalConfig.d);
            currentValue = Eval(values + idx * globalConfig.d);

            if (currentValue < bestValue)
            {
                bestValue = currentValue;
                bestbit = bitflip;
                
            } else if (hiprand_uniform(&states[idx]) < exp(-fabs(currentValue - bestValue) / T)){
                bestValue = currentValue;
                bestbit = bitflip;
                
            }
            bitstr[bitflip] = !bitstr[bitflip];
        }
        

    } while (changeCount < maxAttempts && counter < 10 * maxAttempts);

    candidates[idx] = bestValue;
}

std::vector<double> launch(const Config& config) {

    bool* bitstr;
    double* candidates;
    double* realValues;
    hiprandState* states;
    std::vector<double> result(config.it);

    // Allocate device memory
    hipMalloc(&bitstr, sizeof(bool) * config.bits * config.it);
    hipMalloc(&candidates, sizeof(double) * config.it);
    hipMalloc(&states, sizeof(hiprandState) * config.it);
    hipMalloc(&realValues, sizeof(double) * config.it * config.d);
    hipMemcpyToSymbol(HIP_SYMBOL(globalConfig), &config, sizeof(Config));


    // Launch kernel
    Init_states << < config.blocks, config.threads >> > (states, std::random_device{}());
    InitBitstring << < config.blocks, config.threads >> > (states, bitstr);
    GenRealValues << < config.blocks, config.threads >> > (bitstr, realValues);
    EvalFitness << < config.blocks, config.threads >> > (realValues, candidates);


    switch (globalConfig.strat)
    {
    case improvment::Firstimprov:
        HillClimbFirstImpr << < config.blocks, config.threads >> > (bitstr, realValues, candidates);
        break;
    case improvment::Bestimprov:
        HillClimbBestImpr << < config.blocks, config.threads >> > (bitstr, realValues, candidates);
        break;
    case improvment::Worstimprov:
        HillClimbWorstImpr << < config.blocks, config.threads >> > (bitstr, realValues, candidates);
        break;
    case improvment::Annealing:
        Annealing << < config.blocks, config.threads >> > (bitstr, realValues, candidates, states);
        break;
    default:
        break;
    }


    // Copy result back to host
    hipMemcpy(result.data(), candidates, sizeof(double) * config.it, hipMemcpyDeviceToHost);


    // Clean up device memory
    hipFree(bitstr);
    hipFree(candidates);
    hipFree(states);
    hipFree(realValues);

    return result;
}